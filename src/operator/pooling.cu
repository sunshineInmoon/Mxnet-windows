/*!
 * Copyright (c) 2015 by Contributors
 * \file pooling.cu
 * \brief
 * \author Bing Xu
*/

#include "./pooling-inl.h"
#if MXNET_USE_CUDNN == 1
#include "./cudnn_pooling-inl.h"
#endif  // MXNET_USE_CUDNN

namespace mxnet {
namespace op {
template<>
Operator *CreateOp<gpu>(PoolingParam param, int dtype,
                   std::vector<TShape> *in_shape,
                   std::vector<TShape> *out_shape) {
  Operator *op = NULL;
#if MXNET_USE_CUDNN == 1
  MSHADOW_REAL_TYPE_SWITCH(dtype, DType, {
    switch (param.pool_type) {
      case pool_enum::kMaxPooling:
        op = new CuDNNPoolingOp<DType>(param);
        break;
      case pool_enum::kAvgPooling:
        op = new CuDNNPoolingOp<DType>(param);
        break;
      case pool_enum::kSumPooling:
        LOG(WARNING) << "Sum pooling is not supported by cudnn, MxNet sum pooling is applied.";
        op = new PoolingOp<gpu, mshadow::red::sum, DType>(param);
        break;
      default:
        LOG(FATAL) << "unknown pooling type";
        return NULL;
    }
  });
#else
  MSHADOW_REAL_TYPE_SWITCH(dtype, DType, {
    switch (param.pool_type) {
      case pool_enum::kMaxPooling:
        op = new PoolingOp<gpu, mshadow::red::maximum, DType>(param);
        break;
      case pool_enum::kAvgPooling:
        op = new PoolingOp<gpu, mshadow::red::sum, DType>(param);
        break;
      case pool_enum::kSumPooling:
        op = new PoolingOp<gpu, mshadow::red::sum, DType>(param);
        break;
      default:
        LOG(FATAL) << "unknown pooling type";
        return NULL;
    }
  });
#endif  // MXNET_USE_CUDNN
  return op;
}

}  // namespace op
}  // namespace mxnet

