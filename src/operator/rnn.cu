/*!
 * Copyright (c) 2015 by Contributors
 * \file rnn.cu
 * \brief
 * \author Sebastian Bodenstein
*/

#include "./rnn-inl.h"
#include <algorithm>
#if MXNET_USE_CUDNN == 1 && CUDNN_MAJOR == 5
#include "./cudnn_rnn-inl.h"
#endif  // MXNET_USE_CUDNN && CUDNN_MAJOR

namespace mxnet {
namespace op {
template<>
Operator* CreateOp<gpu>(RNNParam param, int dtype) {
  Operator *op = NULL;
#if MXNET_USE_CUDNN == 1 && CUDNN_MAJOR == 5
  MSHADOW_REAL_TYPE_SWITCH(dtype, DType, {
    op = new CuDNNRNNOp<DType>(param);
  })
#else
  LOG(FATAL) << "RNN is only available for cuDNN at the moment.";
#endif  // MXNET_USE_CUDNN && CUDNN_MAJOR
  return op;
}

}  // namespace op
}  // namespace mxnet
