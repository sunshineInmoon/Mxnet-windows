/*!
 * Copyright (c) 2015 by Contributors
 * \file dropout.cc
 * \brief
 * \author Bing Xu
*/

#include "./dropout-inl.h"

namespace mxnet {
namespace op {
template<>
Operator *CreateOp<gpu>(DropoutParam param, int dtype) {
  Operator *op = NULL;
  MSHADOW_REAL_TYPE_SWITCH(dtype, DType, {
    op = new DropoutOp<gpu, DType>(param);
  });
  return op;
}
}  // namespace op
}  // namespace mxnet


