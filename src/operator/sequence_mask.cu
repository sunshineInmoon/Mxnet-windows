#include "hip/hip_runtime.h"
/*!
 * Copyright (c) 2015 by Contributors
 * \file sequence_mask.cu
 * \brief
 * \author Sebastian Bodenstein
*/

#include "./sequence_mask-inl.h"


namespace mshadow {
namespace cuda {

////////////////////////////////////////////////////////////////////////////////
// Cross-Entropy loss
template<int n_bits, typename DType>
__global__ void SequenceMaskKernel(Tensor<gpu, 3, DType> dst,
                    const Tensor<gpu, 1, DType> lengths) {
  const index_t smax = dst.size(0);
  const index_t bmax = lengths.size(1);
  const index_t nmax = dst.size(2);
  unsigned int batch = threadIdx.x + blockIdx.x * blockDim.x;

  // early return if out of bounds
  if (batch >= bmax)
    return;

  // loop over batches
    for (index_t s = lengths[batch]; s < smax; ++s)
      for (index_t r = 0; r < nmax; ++r)
      dst[s][batch][r] = 0.;
}

////////////////////////////////////////////////////////////////////////////////

template<typename DType>
inline void SequenceMask(const Tensor<gpu, 3, DType> &dst,
                         const Tensor<gpu, 1, DType> &lengths) {
  dim3 dimBlock(kBaseThreadNum);
  dim3 dimGrid(dst.size(1));
  CheckLaunchParam(dimGrid, dimBlock, "SequenceMask");
  hipStream_t stream = Stream<gpu>::GetStream(dst.stream_);
  SequenceMaskKernel<kBaseThreadBits, DType><<<dimGrid, dimBlock, 0, stream>>>(dst, lengths);
}

////////////////////////////////////////////////////////////////////////////////
}  // namespace cuda

template<typename DType>
inline void SequenceMask(Tensor<gpu, 3, DType> dst,
                   const Tensor<gpu, 1, DType> &lengths) {
  cuda::SequenceMask(dst, lengths);
}

}  // namespace mshadow

////////////////////////////////////////////////////////////////////////////////

namespace mxnet {
namespace op {
template <> Operator *CreateOp<gpu>(SequenceMaskParam param, int dtype) {
  Operator *op = NULL;
  MSHADOW_REAL_TYPE_SWITCH(dtype, DType,
                           { op = new SequenceMaskOp<gpu, DType>(param); })
  return op;
}

}  // namespace op
}  // namespace mxnet
